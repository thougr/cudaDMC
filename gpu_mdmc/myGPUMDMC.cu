#include "hip/hip_runtime.h"
//
// Created by hougr t on 2024/1/28.
//

#include <chrono>
#include "myGPUMDMC.cuh"

#include "vtkArrayDispatch.h"
#include "vtkCellArray.h"
#include "vtkCharArray.h"
#include "vtkDataArrayRange.h"
#include "vtkDoubleArray.h"
#include "vtkFloatArray.h"
#include "vtkImageTransform.h"
#include "vtkIncrementalPointLocator.h"
#include "vtkInformation.h"
#include "vtkInformationVector.h"
#include "vtkIntArray.h"
#include "vtkLongArray.h"
#include "vtkMarchingCubesTriangleCases.h"
#include "vtkMath.h"
#include "vtkMergePoints.h"
#include "vtkObjectFactory.h"
#include "vtkPointData.h"
#include "vtkPolyData.h"
#include "vtkShortArray.h"
#include "vtkStreamingDemandDrivenPipeline.h"


vtkStandardNewMacro(myGPUMDMC);
#define GLM_FORCE_CUDA
#include "../glm/glm.hpp"
#include "../table.cuh"
#include "../common.cuh"
#include "../Octree.cuh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

__host__ __device__ glm::vec3 convertToRelative(Direction dir) {
    glm::vec3 v(0, 0, 0);
    switch (dir) {
        case Direction::BOTTOM:
            v = {0, 0, -1};
            break;
        case Direction::TOP:
            v = {0, 0, 1};
            break;
        case Direction::FRONT:
            v = {0, -1, 0};
            break;
        case Direction::BACK:
            v = {0, 1, 0};
            break;
        case Direction::LEFT:
            v = {-1, 0, 0};
            break;
        case Direction::RIGHT:
            v = {1, 0, 0};
            break;
        default:
            break;
    }
    return v;
}


template<class T, class B>
__host__ __device__ Octree<T,B> * getNeighbour(VoxelsData<T> *voxelsData, Octree<T,B> *leafNodes, glm::vec3 v, glm::u32vec3 origin) {
    glm::vec3 neighbourPos = v + glm::vec3(origin);
    if (neighbourPos.x >= voxelsData->cubeDims.x || neighbourPos.y >= voxelsData->cubeDims.y || neighbourPos.z >= voxelsData->cubeDims.z) {
        return nullptr;
    }
    if (neighbourPos.x < 0 || neighbourPos.y < 0 || neighbourPos.z < 0) {
        return nullptr;
    }
    return &leafNodes[position2Index(neighbourPos, voxelsData->cubeDims)];
}

template<class T, class B>
__global__ void generateLeafNodes(VoxelsData<T> *voxelsData, Octree<T,B> *leafNodes, int size, int depth, double isovalue, OctreeRepresentative *representatives) {
    unsigned stride=gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
//    unsigned blockId = (gridDim.x * gridDim.y * blockIdx.z) + (gridDim.x * blockIdx.y) + blockIdx.x;
//    unsigned offset = (blockId * (blockDim.x * blockDim.y * blockDim.z)) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for (int i = offset; i < size;i += stride) {
        leafNodes[i] = Octree<T,B>();
        auto &leaf = leafNodes[i];

        leaf.type = OctreeNodeType::Node_Leaf;
        leaf.height = 0;
        leaf.depth = depth;
        leaf.isoValue = isovalue;
        unsigned x = i % voxelsData->cubeDims.x;
        unsigned y = (i / voxelsData->cubeDims.x) % voxelsData->cubeDims.y;
        unsigned z = i / (voxelsData->cubeDims.x * voxelsData->cubeDims.y);
        unsigned index = (x % 2) + ((y % 2) << 1) + ((z % 2) << 2);
        glm::u32vec3 origin;
        origin.x = x;
        origin.y = y;
        origin.z = z;
        glm::u32vec3 regionSize;
        regionSize.x = 1;
        regionSize.y = 1;
        regionSize.z = 1;
        Region region = {.origin = origin, .size = regionSize, .voxelsCnt = 1, .conceptualSize = regionSize};
        leaf.region = region;
        leaf.index = index;
        auto p2Index = position2Index(region.origin, voxelsData->dims);
        leaf.maxScalar = voxelsData->scalars[p2Index];
        leaf.minScalar = leaf.maxScalar;
        for (int j = 0; j < 8; j++) {
            glm::u32vec3 verticesPos = region.origin + device_localVerticesPos[j];
//            verticesPos.x += localVerticesPos[j].x;
//            verticesPos.y += localVerticesPos[j].y;
//            verticesPos.z += localVerticesPos[j].z;
            auto scalar = voxelsData->scalars[position2Index(verticesPos, voxelsData->dims)];
            double n[3];
            vtkMarchingCubesComputePointGradient(verticesPos, voxelsData->scalars, voxelsData->dims, voxelsData->dims[0] * voxelsData->dims[1], n);
            leaf.normal[j] = {n[0], n[1], n[2]};
            if (scalar > leaf.maxScalar) {
                leaf.maxScalar = scalar;
            }
            if (scalar < leaf.minScalar) {
                leaf.minScalar = scalar;
            }
            leaf.scalar[j] = scalar;
            leaf.sign |= (scalar >= isovalue) ? 1 << j : 0;
        }


        for (int j = 0; j < 6; j++) {
            Direction dir = static_cast<Direction>(j);
            glm::vec3 v = convertToRelative(dir);
            glm::vec3 neighbourPos = v + glm::vec3(origin);
//            neighbourPos.x += origin.x;
//            neighbourPos.y += origin.y;
//            neighbourPos.z += origin.z;
            if (neighbourPos.x >= voxelsData->cubeDims.x || neighbourPos.y >= voxelsData->cubeDims.y || neighbourPos.z >= voxelsData->cubeDims.z) {
                leaf.neighbour[j] = nullptr;
                continue;
            }
            if (neighbourPos.x < 0 || neighbourPos.y < 0 || neighbourPos.z < 0) {
                leaf.neighbour[j] = nullptr;
                continue;
            }
            leaf.neighbour[j] = &leafNodes[position2Index(neighbourPos, voxelsData->cubeDims)];
        }

        if (leaf.maxScalar < isovalue || leaf.minScalar > isovalue) {
            leafNodes[i].type = Node_None;
        } else {
            Octree<T,B>::calculateMDCRepresentative(&leaf, &leaf, representatives, i, isovalue, 1);
        }
    }
}

template<class T, class B>
__global__ void generateInternalNodes(VoxelsData<T> *voxelsData, Octree<T,B> *internalNodes, Octree<T,B> *childrenNodes,
                                      int size, int depth, int height, double isovalue,
                                      glm::u32vec3 childrenDims, glm::u32vec3 dims, unsigned regionSize) {
    unsigned stride = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for(int i = offset; i < size;i += stride){
        internalNodes[i] = Octree<T,B>();
        auto &leaf = internalNodes[i];
        leaf.depth = depth;
        leaf.height = height;
        unsigned x = i % dims.x;
        unsigned y = (i / dims.x) % dims.y;
        unsigned z = i / (dims.x * dims.y);
        unsigned index = (x % 2) + ((y % 2) << 1) + ((z % 2) << 2);
        glm::u32vec3 origin = {x, y, z};
        origin *= regionSize;
        glm::u32vec3 maxBound = origin + regionSize;
        maxBound = glm::min(maxBound, voxelsData->dims);
        maxBound -= origin;

        leaf.index = index;
        Region region = {.origin = origin , .size = maxBound, .voxelsCnt = maxBound.x * maxBound.y * maxBound.z, .conceptualSize = {regionSize, regionSize, regionSize}};
        leaf.region = region;
        leaf.type = OctreeNodeType::Node_Internal;
        leaf.isoValue = isovalue;

        T maxScalar = voxelsData->scalars[position2Index(region.origin, voxelsData->dims)];
        T minScalar = maxScalar;
        unsigned childrenSize = regionSize / 2u;
        for (int j = 0; j < 8; j++) {
            glm::u32vec3 relativeOrigin = origin + orderOrigin[j] * childrenSize;
            if (relativeOrigin.x >= voxelsData->cubeDims.x || relativeOrigin.y >= voxelsData->cubeDims.y || relativeOrigin.z >= voxelsData->cubeDims.z) {
                leaf.children[j] = nullptr;
                continue;
            }
            leaf.children[j] = &childrenNodes[position2Index(relativeOrigin / childrenSize, childrenDims)];
            if (leaf.children[j]->type == Node_None) {
                leaf.children[j] = nullptr;
                continue;
            }
            if (leaf.children[j]->maxScalar > maxScalar) {
                maxScalar = leaf.children[j]->maxScalar;
            }
            if (leaf.children[j]->minScalar < minScalar) {
                minScalar = leaf.children[j]->minScalar;
            }
        }
        leaf.maxScalar = maxScalar;
        leaf.minScalar = minScalar;
        if (leaf.maxScalar < isovalue || leaf.minScalar > isovalue) {
            leaf.type = Node_None;
        }
    }
}


template<class T, class B>
__global__ void calculateVerticesCnt(Octree<T,B> *leafNodes, int size, int *verticesCnt) {
    unsigned stride = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for (int i = offset; i < size; i += stride) {
        auto leaf = &leafNodes[i];
        if (leaf->type == Node_None) {
            continue;
        }
        int curCnt = 0;
        OctreeRepresentative *vs[12];
        for (int j = 0; j < 12; j++) {
            auto representative = leaf->representative[j];
            if (representative == nullptr) {
                continue;
            }
            representative = findRepresentative(representative);
            bool isExist = false;
            for (int z = 0; z < curCnt; z++) {
                if (vs[z] == representative) {
                    isExist = true;
                    break;
                }
            }
            if (isExist) {
                continue;
            }
            vs[curCnt++] = representative;
            atomicAdd(verticesCnt, 1);
        }

    }
}


template<class T, class B>
__global__ void generateVerticesIndices(Octree<T,B> *leafNodes, int size, int *verticesIndices, glm::vec3 *vertices) {
    unsigned stride = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for (int i = offset; i < size; i += stride) {
        auto leaf = &leafNodes[i];
        if (leaf->type == Node_None) {
            continue;
        }
        int curCnt = 0;
        OctreeRepresentative *vs[12];
        int indices[12];
        for (int j = 0; j < 12; j++) {
            auto representative = leaf->representative[j];
            if (representative == nullptr) {
                continue;
            }
            representative = findRepresentative(representative);
            bool isExist = false;
            for (int z = 0; z < curCnt; z++) {
                if (vs[z] == representative) {
                    isExist = true;
                    break;
                }
            }
            if (isExist) {
                continue;
            }
            int currIndex = atomicAdd(verticesIndices, 1);
            vs[curCnt] = representative;
            indices[curCnt] = currIndex;
            representative->index = currIndex;
            vertices[currIndex] = representative->position;
            curCnt++;
        }

    }
}
template<class T, class B>
__host__ __device__ bool isAllNodeValid(const Octree<T,B> *nodes[4]) {
    for (int i = 0; i < 4; i++) {
        if (nodes[i] == nullptr) {
            return false;
        }
        if (nodes[i]->type == Node_None) {
            return false;
        }
    }
    return true;
}

template<class T, class B>
__device__ void contourProcessEdge(const Octree<T, B> *root[4], int dir, glm::u32vec3 *newPolys, int triIndex, int useOptimization) {
    int minIndex = 0;
    long long indices[4] = {-1, -1, -1, -1};
    bool flip = false;
    bool signChange[4] = {false, false, false, false};
    OctreeRepresentative *representatives[4];
    int intersections[4] = {0, 0, 0, 0};
    auto minAngle = [&] (const glm::vec3 v0, const glm::vec3 v1, const glm::vec3 v2) -> double
    {
        const float da = glm::distance(v0, v1); // std::sqrt((v1.x - v0.x) * (v1.x - v0.x) + (v1.y - v0.y) * (v1.y - v0.y) + (v1.z - v0.z) * (v1.z - v0.z));
        const float db = glm::distance(v1, v2); // std::sqrt((v2.x - v1.x) * (v2.x - v1.x) + (v2.y - v1.y) * (v2.y - v1.y) + (v2.z - v1.z) * (v2.z - v1.z));
        const float dc = glm::distance(v2, v0); // std::sqrt((v0.x - v2.x) * (v0.x - v2.x) + (v0.y - v2.y) * (v0.y - v2.y) + (v0.z - v2.z) * (v0.z - v2.z));
        const float dA = std::acos((db * db + dc * dc - da * da) / (2 * db * dc));
        const float dB = std::acos((da * da + dc * dc - db * db) / (2 * da * dc));
        const float dC = std::acos((db * db + da * da - dc * dc) / (2 * db * da));

        return min(min(dA, dB), dC);
    };
    for (int i = 0; i < 4; i++) {
        const int edge = device_processEdgeMask[dir][i];
        int c1 = device_edges2Vertices[edge][0];
        int c2 = device_edges2Vertices[edge][1];
        auto m1 = (root[i]->sign >> c1) & 1;
        auto m2 = (root[i]->sign >> c2) & 1;
        auto vertex = root[i]->representative[edge];

        if (m1 ^ m2) {
            signChange[i] = true;
            vertex = findRepresentative(vertex);
            representatives[i] = vertex;
            indices[i] = vertex->index;
            intersections[i] = vertex->edgeIntersection[edge];
        }
        flip = m1;
    }
    auto isAllDifferent = [&] (long long *tris, int len) {
        // more quickly, use unordered_set
//        std::unordered_set<long long> set;
        for (int i = 0; i < 3; i++) {
            if (tris[i] == -1) {
                return false;
            }
        }
        return tris[0] != tris[1] && tris[0] != tris[2] && tris[1] != tris[2];
//        for (int i = 0; i < len; i++) {
//            if (set.find(tris[i]) != set.end() || tris[i] == -1) {
//                return false;
//            }
//            set.insert(tris[i]);
//        }
//        return true;
    };

    auto insertTriangle = [&] (long long verticesId[4], const int indices[4], glm::u32vec3 *newPolys, int index) {
        int tris1[3] = {0, 2, 3};
        int tris2[3] = {0, 1, 2};
        if (intersections[indices[0]] == 2 && intersections[indices[2]] == 2) {
            tris1[1] = 1;
            tris2[0] = 1;
            tris2[1] = 2;
            tris2[2] = 3;
//            tris1 = {0, 1, 3};
//            tris2 = {1, 2, 3};
        }
        {
            long long tris[3];
            int realIndices[3];
            for (int i = 0; i < 3; i++) {
                auto realIndex = indices[tris1[i]];
                realIndices[i] = realIndex;
                tris[i] = verticesId[realIndex];
            }
            if (isAllDifferent(tris, 3)) {
                newPolys[index] = glm::u32vec3(tris[0], tris[1], tris[2]);
            }
        }
        {
            long long tris[3];
            int realIndices[3];
            for (int i = 0; i < 3; i++) {
                auto realIndex = indices[tris2[i]];
                realIndices[i] = realIndex;
                tris[i] = verticesId[realIndex];
            }
            if (isAllDifferent(tris, 3)) {
                newPolys[index + 1] = glm::u32vec3(tris[0], tris[1], tris[2]);
            }
        }
    };

    if (signChange[minIndex]) {
        auto v0 = representatives[0]->position;
        auto v1 = representatives[1]->position;
        auto v2 = representatives[2]->position;
        auto v3 = representatives[3]->position;
        double a1_ = minAngle(v0, v1, v2);
        double a2_ = minAngle(v2, v3, v1);
        const double b1_ = min(a1_, a2_);
        const double b2_ = max(a1_, a2_);
        a1_ = minAngle(v0, v1, v3);
        a2_ = minAngle(v0, v2, v3);
        const double c1_ = min(a1_, a2_);
        const double c2_ = max(a1_, a2_);
//        if (flip) {
//            const int ins[4] = {0, 2, 3, 1};
//            insertTriangle(indices, ins, newPolys);
//
//        } else {
//            const int ins[4] = {0, 1, 3, 2};
//            insertTriangle(indices, ins, newPolys);
//        }

        if (!useOptimization || b1_ < c1_ || (b1_ == c1_ && b2_ <= c2_))
        {
            if (flip) {
                const int ins[4] = {0, 2, 3, 1};
                insertTriangle(indices, ins, newPolys, triIndex);

            } else {
                const int ins[4] = {0, 1, 3, 2};
                insertTriangle(indices, ins, newPolys, triIndex);
            }
        } else {
            if (flip) {
                const int ins[4] = {2, 3, 1, 0};
                insertTriangle(indices, ins, newPolys, triIndex);

            } else {
                const int ins[4] = {2, 0, 1, 3};
                insertTriangle(indices, ins, newPolys, triIndex);
            }
        }
    }
}

template<class T, class B>
__global__ void calculateQuadCnt(VoxelsData<T> *voxelsData, Octree<T,B> *leafNodes, int size, int *quadCnt) {
    unsigned stride = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for (int i = offset; i < size; i += stride) {
        auto leaf = &leafNodes[i];
        if (leaf->type == Node_None) {
            continue;
        }
        const int edgeCheck[3] = {5, 6, 10};
        const int dirs[3] = {1, 0, 2};
        if (leaf->representative[5] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::RIGHT],
                                                    leaf->neighbour[Direction::TOP],
//            getNeighbour(voxelsData, leafNodes, {1, 0, 0}, leaf->region.origin),
//            getNeighbour(voxelsData, leafNodes, {0, 0, 1}, leaf->region.origin),
                                                    getNeighbour(voxelsData, leafNodes, {1, 0, 1}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                atomicAdd(quadCnt, 1);
            }
        }

        if (leaf->representative[6] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::TOP],
                                                    leaf->neighbour[Direction::BACK],
//                                                    getNeighbour(voxelsData, leafNodes, {0, 0, 1}, leaf->region.origin),
//                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 0}, leaf->region.origin),
                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 1}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                atomicAdd(quadCnt, 1);
            }
        }

        if (leaf->representative[10] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::BACK],
                                                    leaf->neighbour[Direction::RIGHT],

//                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 0}, leaf->region.origin),
//                                                    getNeighbour(voxelsData, leafNodes, {1, 0, 0}, leaf->region.origin),

                                                    getNeighbour(voxelsData, leafNodes, {1, 1, 0}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                atomicAdd(quadCnt, 1);
            }
        }
    }
}



template<class T, class B>
__global__ void generateQuad(VoxelsData<T> *voxelsData, Octree<T,B> *leafNodes, int size, glm::u32vec3 *tris, int *triIndex) {
    unsigned stride = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    unsigned blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    unsigned offset = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    for (int i = offset; i < size; i += stride) {
        auto leaf = &leafNodes[i];
        if (leaf->type == Node_None) {
            continue;
        }
        const int edgeCheck[3] = {5, 6, 10};
        const int dirs[3] = {1, 0, 2};
        if (leaf->representative[5] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::RIGHT],
                                                    leaf->neighbour[Direction::TOP],
//            getNeighbour(voxelsData, leafNodes, {1, 0, 0}, leaf->region.origin),
//            getNeighbour(voxelsData, leafNodes, {0, 0, 1}, leaf->region.origin),
                                                    getNeighbour(voxelsData, leafNodes, {1, 0, 1}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                auto index = atomicAdd(triIndex, 2);
                contourProcessEdge(neighbourCheck, dirs[0], tris, index, 1);
            }
        }

        if (leaf->representative[6] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::TOP],
                                                    leaf->neighbour[Direction::BACK],
//                                                    getNeighbour(voxelsData, leafNodes, {0, 0, 1}, leaf->region.origin),
//                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 0}, leaf->region.origin),
                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 1}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                auto index = atomicAdd(triIndex, 2);
                contourProcessEdge(neighbourCheck, dirs[1], tris, index, 1);
            }
        }

        if (leaf->representative[10] != nullptr) {
            const Octree<T,B> *neighbourCheck[4] = {leaf,
                                                    leaf->neighbour[Direction::BACK],
                                                    leaf->neighbour[Direction::RIGHT],

//                                                    getNeighbour(voxelsData, leafNodes, {0, 1, 0}, leaf->region.origin),
//                                                    getNeighbour(voxelsData, leafNodes, {1, 0, 0}, leaf->region.origin),

                                                    getNeighbour(voxelsData, leafNodes, {1, 1, 0}, leaf->region.origin)};
            if (isAllNodeValid(neighbourCheck)) {
                auto index = atomicAdd(triIndex, 2);
                contourProcessEdge(neighbourCheck, dirs[2], tris, index, 1);
            }
        }
    }
}

namespace {

    struct ComputeGradientWorker
    {
        template <class ScalarArrayT>
        __host__ void operator()(ScalarArrayT* scalarsArray, isosurfacesAlgorithm* self, int dims[3],
                        vtkIncrementalPointLocator* locator, vtkDataArray* newScalars, vtkDataArray* newGradients,
                        vtkDataArray* newNormals, vtkCellArray* newPolys, double* values, vtkIdType numValues) const
        {
            int extent[6];
            double value = values[0];

            vtkInformation* inInfo = self->GetExecutive()->GetInputInformation(0, 0);
            inInfo->Get(vtkStreamingDemandDrivenPipeline::WHOLE_EXTENT(), extent);
            using ComponentRef =  typename vtk::detail::SelectValueRange<ScalarArrayT, 1>::type;
            using ArrayType  = decltype(scalarsArray->GetArrayType());
            using OctreeType = Octree<ArrayType, ComponentRef>;
            glm::u32vec3 size =  {dims[0] - 1, dims[1] - 1, dims[2] - 1};
            int height = 0;
            glm::u32vec3 conceptualSize = findLargerClosestPower2Vector(size, height);
            auto numberOfTuples = scalarsArray->GetNumberOfTuples();
            auto scalars = vtk::DataArrayValueRange<1>(scalarsArray);

            VoxelsData<ArrayType> voxelsData = {.scalars = new ArrayType[numberOfTuples],
                    .cubeDims = size, .dims = {dims[0], dims[1], dims[2]}, .conceptualDims = conceptualSize };
            for (int i = 0; i < numberOfTuples; i++) {
                voxelsData.scalars[i] = scalars[i];
            }

            VoxelsData<ArrayType> *deviceData;
            hipMallocManaged(&deviceData, sizeof(VoxelsData<ArrayType>));
            hipMemcpy(deviceData, &voxelsData, sizeof(VoxelsData<ArrayType>), hipMemcpyHostToDevice);
            ArrayType *deviceScalars;
            hipMallocManaged(&deviceScalars, sizeof(ArrayType) * numberOfTuples);
            hipMemcpy(deviceScalars, voxelsData.scalars, sizeof(ArrayType) * numberOfTuples, hipMemcpyHostToDevice);
            // set deviceScalars to deviceData.scalars
            hipMemcpy(&(deviceData->scalars), &deviceScalars, sizeof(ArrayType *), hipMemcpyHostToDevice);
//            Region region = {.origin = {0, 0, 0}, .size = size,
//                    .voxelsCnt = size.x * size.y * size.z, .conceptualSize = conceptualSize};
            // open a file and read tree

            std::cout << voxelsData.scalars[0] << std::endl;
            std::cout << voxelsData.dims[0] << " " << voxelsData.dims[1] << " " << voxelsData.dims[2] << std::endl;
            std::cout << voxelsData.conceptualDims[0] << " " << voxelsData.conceptualDims[1] << " " << voxelsData.conceptualDims[2] << std::endl;
            std::cout << voxelsData.cubeDims[0] << " " << voxelsData.cubeDims[1] << " " << voxelsData.cubeDims[2] << std::endl;

            dim3 grid(32,32);
            dim3 block(32,32);
//    dim3 block={16, 8, 8};
//    dim3 grid={conceptualSize.x / block.x, conceptualSize.y / block.y, conceptualSize.z / block.z};
            // better grid and block for v100
            auto begin = std::chrono::high_resolution_clock::now();;
            auto start = std::chrono::high_resolution_clock::now();;
            OctreeType *leafNodes = nullptr;
            OctreeRepresentative *octreeRepresentatives = nullptr;
            int leafNum = size.x * size.y * size.z;
            {
//                Region region = {.origin = {0, 0, 0}, .size = size,
//                        .voxelsCnt = size.x * size.y * size.z, .conceptualSize = conceptualSize};
                long long nByte = 1ll * sizeof(OctreeType) * leafNum;
                hipMallocManaged((void**)&leafNodes, nByte);
                hipMemset(leafNodes, 0, nByte);
                nByte = 4ll * sizeof(OctreeRepresentative) * leafNum;
                hipMallocManaged((void**)&octreeRepresentatives, nByte);
                generateLeafNodes<<<grid, block>>>(deviceData, leafNodes, leafNum, height, value, octreeRepresentatives);
                auto error = hipDeviceSynchronize();
                // print the error
                if (error != hipSuccess) {
                    std::cout << hipGetErrorString(error) << std::endl;
                }
            }

            {
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<double> duration = end - start;
                double seconds = duration.count();
                start = end;
                std::cout << "构建叶子节点: " << seconds << " 秒" << std::endl;

            }


            OctreeType **everyHeightNodes = new OctreeType*[height+1];
            everyHeightNodes[0] = leafNodes;
            unsigned regionSize = 1;
            glm::u32vec3 childrenSize = size;
            for (int h = 1; h <= height; h++) {
                OctreeType *nodes = nullptr;
                regionSize *= 2;
                // roundUp
                glm::u32vec3 curSize = (childrenSize + 1u) / 2u;
                int nodeNums = curSize.x * curSize.y * curSize.z;
                long long nByte = 1ll * sizeof(OctreeType) * nodeNums;
                hipMallocManaged((void**)&nodes, nByte);
                hipMemset(nodes, 0, nByte);
                generateInternalNodes<<<grid, block>>>(deviceData, nodes, everyHeightNodes[h-1], nodeNums, height - h, h, value, childrenSize, curSize, regionSize);
                hipDeviceSynchronize();
                everyHeightNodes[h] = nodes;
                childrenSize = curSize;
            }
            {
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<double> duration = end - start;
                double seconds = duration.count();
                start = end;
                std::cout << "构建中间节点: " << seconds << " 秒" << std::endl;
            }

            OctreeType *root = everyHeightNodes[height];


            OctreeType *isoTree = root;


// isoTree = OctreeType::simplifyOctree(isoTree, 1.1f);
//            OctreeType::clusterCell(isoTree, 1000.1f, 1);
            auto adaptiveThreshold = self->GetAdaptiveThreshold();
//            if (adaptiveThreshold != 0) {
//                OctreeType::clusterCell(isoTree, adaptiveThreshold, 1);
//            }
//            OctreeType::generateVerticesIndices(isoTree, locator, newScalars);
            int *d_count;
            hipMallocManaged(&d_count, sizeof(int ));
            hipMemset(d_count, 0, sizeof(int ));
            calculateVerticesCnt<<<grid, block>>>(leafNodes, leafNum, d_count);
            hipDeviceSynchronize();
            std::cout << "顶点个数:" << d_count[0] << std::endl;
            glm::vec3 *vertices;
            int *vertexIndex;
            hipMallocManaged(&vertexIndex, sizeof(int));
            hipMemset(vertexIndex, 0, sizeof(int));
            hipMallocManaged(&vertices, 1ll * sizeof(glm::vec3) * d_count[0]);
            generateVerticesIndices<<<grid, block>>>(leafNodes, leafNum, vertexIndex, vertices);
            hipDeviceSynchronize();
            std::cout << "数组顶点个数:" << vertexIndex[0] << std::endl;
            {
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<double> duration = end - start;
                double seconds = duration.count();
                start = end;
                std::cout << "生成顶点: " << seconds << " 秒" << std::endl;
            }


            int *quadCnt;
            hipMallocManaged(&quadCnt, sizeof(int));
            hipMemset(quadCnt, 0, sizeof(int));
            calculateQuadCnt<<<grid, block>>>(deviceData, leafNodes, leafNum, quadCnt);
            hipDeviceSynchronize();
            std::cout << "三角面个数:" << quadCnt[0] * 2 << std::endl;
            glm::u32vec3 *tris;
            int *trisIndex;
            hipMallocManaged(&trisIndex, sizeof(int));
            hipMemset(trisIndex, 0, sizeof(int));
            hipMallocManaged(&tris, 2ll * sizeof(glm::u32vec3) * quadCnt[0]);
            generateQuad<<<grid, block>>>(deviceData, leafNodes, leafNum, tris, trisIndex);
            hipDeviceSynchronize();
            std::cout << "数组三角面个数:" << trisIndex[0] << std::endl;


//            OctreeType::contourCellProc(isoTree, newPolys, 1);


//            OctreeType::destroyOctree(isoTree);


//            Octree<ArrayType,ComponentRef>::destroyOctree(allDataTree);
//            Octree<ScalarArrayT>::buildOctree()
            {
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<double> duration = end - start;
                double seconds = duration.count();
                start = end;
                std::cout << "生成三角面: " << seconds << " 秒" << std::endl;
            }
            {
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<double> duration = end - begin;
                double seconds = duration.count();
                std::cout << "总时间: " << seconds << " 秒" << std::endl;
            }

            std::unordered_map<int, vtkIdType > vertexMap;
            for (int i = 0; i < vertexIndex[0]; i++) {
                vtkIdType id;
                double p[3];
                for (int j = 0; j < 3; j++) {
                    p[j] = vertices[i][j];
                }
                if (locator->InsertUniquePoint(p, id)) {
                    newScalars->InsertTuple(id, &value);
                }
                vertexMap[i] = id;
            }
            for (int i = 0; i < trisIndex[0]; i++) {
                vtkIdType ids[3];
                for (int j = 0; j < 3; j++) {
                    ids[j] = vertexMap[tris[i][j]];
                }
                newPolys->InsertNextCell(3, ids);
            }

            hipFree(deviceData);
            delete voxelsData.scalars;
            // free the everyHeightNodes
            for (int i = 0; i <= height; i++) {
                hipFree(everyHeightNodes[i]);
            }
            delete []everyHeightNodes;
            hipFree(d_count);
            hipFree(vertices);
            hipFree(vertexIndex);
            hipFree(quadCnt);
            hipFree(tris);
            hipFree(trisIndex);
        }
    };



}

void myGPUMDMC::process(vtkDataArray* scalarsArray, isosurfacesAlgorithm* self, int dims[3],
                           vtkIncrementalPointLocator* locator, vtkDataArray* newScalars, vtkDataArray* newGradients,
                           vtkDataArray* newNormals, vtkCellArray* newPolys, double* values, vtkIdType numValues) {

    using Dispatcher = vtkArrayDispatch::Dispatch;
    ComputeGradientWorker worker;
    if (!Dispatcher::Execute(scalarsArray, worker, this, dims, this->Locator, newScalars, newGradients,
                             newNormals, newPolys, values, numValues)) { // Fallback to slow path for unknown arrays:
        std::cout << "Fallback to slow path for unknown arrays" << std::endl;
        worker(scalarsArray, this, dims, this->Locator, newScalars, newGradients, newNormals, newPolys,
               values, numValues);
    }
}

bool myGPUMDMC::supportAdaptiveMeshing() {
    return true;
}
