#include "hip/hip_runtime.h"
#include "DualMarchingCubes.h"

// CUDA
#include "hip/hip_runtime_api.h"
#include "CTimer.h"
#include "QualityMeasure.h"
#include "QuadrilateralHashTable.h"
#include "HalfedgeHashTable.h"
#include "EdgeHashTable.h"
#include "VertexHashTable.h"
#include "Vertices.h"
#include "Triangles.h"
#include "Quadrilaterals.h"
#include "Edges.h"
#include "Halfedges.h"
#include "HalfedgeVertices.h"
#include "HalfedgeFaces.h"
#include "MarchingCubesLookupTables.h"
#include "CellIntersection.h"
#include "VertexMap.h"
#include "QuadrilateralMap.h"
#include "HalfedgeMesh.h"
#include "MeshSimplification.h"
#include "FaceColoring.h"
#include "EstimateElementQuality.h"
#include "VertexValence.h"

// Thrust
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>



// type aliases
// Introduce convenient aliases here
using namespace p_mc;
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using ullong = unsigned long long;
using UGrid = p_mc::DualMarchingCubes::UGrid;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      CUDA GLOBAL FUNCTIONS
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      Compute Element Quality and Generate best triangle mesh out of a quadrilateral mesh
//      Use the MaxMin angle criterion
//      Compute triangle angles based on cosine rule
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void quadrilateral_to_triangle(Quadrilaterals q_, Vertices v_, Triangles t_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q)
        return;
    // get vertices
    const int v0 = q_.quadrilaterals[tid].x;
    const int v1 = q_.quadrilaterals[tid].y;
    const int v2 = q_.quadrilaterals[tid].z;
    const int v3 = q_.quadrilaterals[tid].w;
    const float3 p0 = v_.vertices[v0];
    const float3 p1 = v_.vertices[v1];
    const float3 p2 = v_.vertices[v2];
    const float3 p3 = v_.vertices[v3];

    float a1_ = t_.minAngle(p0, p1, p2);
    float a2_ = t_.minAngle(p0, p2, p3);
    float b1_ = fminf(a1_, a2_);
    float b2_ = fmaxf(a1_, a2_);
    a1_ = t_.minAngle(p1, p3, p0);
    a2_ = t_.minAngle(p1, p2, p3);
    float c1_ = fminf(a1_, a2_);
    float c2_ = fmaxf(a1_, a2_);

    if (b1_ < c1_ || (b1_ == c1_ && b2_ <= c2_))
    {
        t_.addTriangle(2 * tid, v1, v3, v0);
        t_.addTriangle(2 * tid + 1, v1, v2, v3);
    }
    else
    {
        t_.addTriangle(2 * tid, v0, v1, v2);
        t_.addTriangle(2 * tid + 1, v0, v2, v3);
    }
    atomicAdd(t_.t_size, 2);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// compute quality measure for a triangle mesh
__global__ void mean_ratio_measure(Triangles t_, Vertices v_, QualityMeasure q_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= t_.nr_t)
        return;
    // collect triangle vertices
    const int v0 = t_.triangles[tid].x;
    const int v1 = t_.triangles[tid].y;
    const int v2 = t_.triangles[tid].z;
    const float3 p0 = v_.vertices[v0];
    const float3 p1 = v_.vertices[v1];
    const float3 p2 = v_.vertices[v2];
    // compute mean ratio quality measure
    q_.mean_ratio(tid, p0, p1, p2);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize hash table for quadrilaterals
__global__ void init_quadrilateral_hashtable(QuadrilateralHashTable ht_)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= ht_.t_size) return;
    ht_.init(tid);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize hash table for vertices, this is a redundant data structure, just to differentiate
__global__ void init_vertex_hashtable(VertexHashTable ht_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= ht_.t_size) return;
    ht_.init(tid);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      DUAL MARCHING CUBES
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// count number of vertices, which is used later to estimate number of faces and allocate memory for buffers
__global__ void count_dmc(const float i0, const uint t_size, UGrid ugrid, CellIntersection c_, int* aCnt)
{
    // use a 1d grid
    const int gl_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_size <= gl_index)
        return;

    const int i_index = ugrid.i_index(gl_index);
    const int j_index = ugrid.j_index(gl_index);
    const int k_index = ugrid.k_index(gl_index);
    if (i_index >= (ugrid.idim - 1) || j_index >= (ugrid.jdim - 1) || k_index >= (ugrid.kdim - 1))
    {
        return;
    }

    // scalar values at vertices
    float u[8];
    u[0] = ugrid(i_index, j_index, k_index);
    u[1] = ugrid(i_index + 1, j_index, k_index);
    u[2] = ugrid(i_index, j_index + 1, k_index);
    u[3] = ugrid(i_index + 1, j_index + 1, k_index);
    u[4] = ugrid(i_index, j_index, k_index + 1);
    u[5] = ugrid(i_index + 1, j_index, k_index + 1);
    u[6] = ugrid(i_index, j_index + 1, k_index + 1);
    u[7] = ugrid(i_index + 1, j_index + 1, k_index + 1);

    //
    uchar i_case{ 0 };
    i_case = i_case + ((uint)(u[0] >= i0));
    i_case = i_case + ((uint)(u[1] >= i0)) * 2;
    i_case = i_case + ((uint)(u[2] >= i0)) * 4;
    i_case = i_case + ((uint)(u[3] >= i0)) * 8;
    i_case = i_case + ((uint)(u[4] >= i0)) * 16;
    i_case = i_case + ((uint)(u[5] >= i0)) * 32;
    i_case = i_case + ((uint)(u[6] >= i0)) * 64;
    i_case = i_case + ((uint)(u[7] >= i0)) * 128;

    if (i_case == 0 || i_case == 255)
        return;
    // intersect cell
    int nr_v = c_.countMCPolygons(i0, i_case, u);
    if (nr_v > 0) atomicAdd(aCnt, nr_v);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Hybrid version
//  - ambiguous cases are processed without lookup table
//  - unambiguous cases are processed with the lookup table containing the MC polygons
__global__ void dual_mc(const float i0, const uint t_size, UGrid ugrid, CellIntersection c_, QuadrilateralHashTable ht_, Vertices v_)
{
    // use a 1d grid
    const int gl_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_size <= gl_index)
        return;

    const int i_index = ugrid.i_index(gl_index);
    const int j_index = ugrid.j_index(gl_index);
    const int k_index = ugrid.k_index(gl_index);
    if (i_index >= (ugrid.idim - 1) || j_index >= (ugrid.jdim - 1) || k_index >= (ugrid.kdim - 1))
    {
        return;
    }

    // scalar values at vertices
    float u[8];
    u[0] = ugrid(i_index, j_index, k_index);
    u[1] = ugrid(i_index + 1, j_index, k_index);
    u[2] = ugrid(i_index, j_index + 1, k_index);
    u[3] = ugrid(i_index + 1, j_index + 1, k_index);
    u[4] = ugrid(i_index, j_index, k_index + 1);
    u[5] = ugrid(i_index + 1, j_index, k_index + 1);
    u[6] = ugrid(i_index, j_index + 1, k_index + 1);
    u[7] = ugrid(i_index + 1, j_index + 1, k_index + 1);

    //
    uchar i_case{ 0 };
    i_case = i_case + ((uint)(u[0] >= i0));
    i_case = i_case + ((uint)(u[1] >= i0)) * 2;
    i_case = i_case + ((uint)(u[2] >= i0)) * 4;
    i_case = i_case + ((uint)(u[3] >= i0)) * 8;
    i_case = i_case + ((uint)(u[4] >= i0)) * 16;
    i_case = i_case + ((uint)(u[5] >= i0)) * 32;
    i_case = i_case + ((uint)(u[6] >= i0)) * 64;
    i_case = i_case + ((uint)(u[7] >= i0)) * 128;

    if (i_case == 0 || i_case == 255)
        return;
    // intersect cell
    c_.slice(i0, i_case, i_index, j_index, k_index, u, ugrid, ht_, v_);
    //c_.sliceQ(i0, i_case, i_index, j_index, k_index, u, ugrid, ht_, v_);
    //c_.sliceP(i0, i_case, i_index, j_index, k_index, u, ugrid, ht_, v_);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      Standard MARCHING CUBES
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// count number of triangles, use an atomic counter
__global__ void count_mc(const float i0, const uint t_size, UGrid ugrid, MarchingCubesLookupTables l_tables, int* aCnt)
{
    // use a 1d grid
    const int gl_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (gl_index >= t_size)
        return;

    const int i_index = ugrid.i_index(gl_index);
    const int j_index = ugrid.j_index(gl_index);
    const int k_index = ugrid.k_index(gl_index);
    if (i_index >= (ugrid.idim - 1) || j_index >= (ugrid.jdim - 1) || k_index >= (ugrid.kdim - 1))
    {
        return;
    }

    // scalar values at vertices
    float u[8];
    u[0] = ugrid(i_index, j_index, k_index);
    u[1] = ugrid(i_index + 1, j_index, k_index);
    u[2] = ugrid(i_index, j_index + 1, k_index);
    u[3] = ugrid(i_index + 1, j_index + 1, k_index);
    u[4] = ugrid(i_index, j_index, k_index + 1);
    u[5] = ugrid(i_index + 1, j_index, k_index + 1);
    u[6] = ugrid(i_index, j_index + 1, k_index + 1);
    u[7] = ugrid(i_index + 1, j_index + 1, k_index + 1);

    // compute case
    uchar i_case{ 0 };
    i_case = i_case + ((uint)(u[0] >= i0));
    i_case = i_case + ((uint)(u[1] >= i0)) * 2;
    i_case = i_case + ((uint)(u[2] >= i0)) * 4;
    i_case = i_case + ((uint)(u[3] >= i0)) * 8;
    i_case = i_case + ((uint)(u[4] >= i0)) * 16;
    i_case = i_case + ((uint)(u[5] >= i0)) * 32;
    i_case = i_case + ((uint)(u[6] >= i0)) * 64;
    i_case = i_case + ((uint)(u[7] >= i0)) * 128;

    if (i_case == 0 || i_case == 255)
        return;

    // count number of triangles
    int nr_t{ 0 };
    for (int t = 0; t < 16; t += 3)
    {
        const int index = i_case * 16 + t;
        const int i0 = static_cast<int>(l_tables.t_pattern[index]);
        if (i0 == -1)
            break;
        nr_t++;
    }
    if (nr_t > 0) atomicAdd(aCnt, nr_t);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// standard marching cubes
__global__ void standard_mc(const float i0, const uint t_size, UGrid ugrid, MarchingCubesLookupTables l_tables, VertexHashTable ht_, Vertices v_, Triangles t_)
{
    // use a 1d grid
    const int gl_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (gl_index >= t_size)
        return;

    const int i_index = ugrid.i_index(gl_index);
    const int j_index = ugrid.j_index(gl_index);
    const int k_index = ugrid.k_index(gl_index);
    if (i_index >= (ugrid.idim - 1) || j_index >= (ugrid.jdim - 1) || k_index >= (ugrid.kdim - 1))
    {
        return;
    }

    // scalar values at vertices
    float u[8];
    u[0] = ugrid(i_index, j_index, k_index);
    u[1] = ugrid(i_index + 1, j_index, k_index);
    u[2] = ugrid(i_index, j_index + 1, k_index);
    u[3] = ugrid(i_index + 1, j_index + 1, k_index);
    u[4] = ugrid(i_index, j_index, k_index + 1);
    u[5] = ugrid(i_index + 1, j_index, k_index + 1);
    u[6] = ugrid(i_index, j_index + 1, k_index + 1);
    u[7] = ugrid(i_index + 1, j_index + 1, k_index + 1);

    // compute case
    uchar i_case{ 0 };
    i_case = i_case + ((uint)(u[0] >= i0));
    i_case = i_case + ((uint)(u[1] >= i0)) * 2;
    i_case = i_case + ((uint)(u[2] >= i0)) * 4;
    i_case = i_case + ((uint)(u[3] >= i0)) * 8;
    i_case = i_case + ((uint)(u[4] >= i0)) * 16;
    i_case = i_case + ((uint)(u[5] >= i0)) * 32;
    i_case = i_case + ((uint)(u[6] >= i0)) * 64;
    i_case = i_case + ((uint)(u[7] >= i0)) * 128;

    if (i_case == 0 || i_case == 255)
        return;

    // compute cell intersection
    // table listing end vertices of an edge
    const unsigned char l_edges_[12]{ 16, 49, 50, 32, 84, 117, 118, 100, 64, 81, 115, 98 };
    const ushort e_ = l_tables.e_pattern[i_case];
    float3 n[8];
    ugrid.gradient(n, u, i_index, j_index, k_index);
    ushort flag{ 1 };
    int v_addr[12];
    for (int e = 0; e < 12; e++)
    {
        v_addr[e] = -1;
        if (flag & e_)
        {
            const int e_id = ugrid.e_glIndex(e, i_index, j_index, k_index);
            // check if vertex was already generated
            const bool v_flag = ht_.addVertex(e_id, v_addr, e);
            if (!v_flag)
            {
                // create vertex
                const int v0 = (l_edges_[e] & 0xF);
                const int v1 = (l_edges_[e] >> 4) & 0xF;
                const float l = (i0 - u[v0]) / (u[v1] - u[v0]);
                const float x0 = ugrid.x0 + (i_index + (v0 & 0x1)) * ugrid.dx;
                const float y0 = ugrid.y0 + (j_index + ((v0 & 0x2) >> 1)) * ugrid.dy;
                const float z0 = ugrid.z0 + (k_index + ((v0 & 0x4) >> 2)) * ugrid.dz;
                const float x1 = ugrid.x0 + (i_index + (v1 & 0x1)) * ugrid.dx;
                const float y1 = ugrid.y0 + (j_index + ((v1 & 0x2) >> 1)) * ugrid.dy;
                const float z1 = ugrid.z0 + (k_index + ((v1 & 0x4) >> 2)) * ugrid.dz;
                float3 ei = make_float3(0, 0, 0);
                float3 ni = make_float3(0, 0, 0);
                ei.x = x0 + l * (x1 - x0);
                ei.y = y0 + l * (y1 - y0);
                ei.z = z0 + l * (z1 - z0);
                ni.x = n[v0].x + l * (n[v1].x - n[v0].x);
                ni.y = n[v0].y + l * (n[v1].y - n[v0].y);
                ni.z = n[v0].z + l * (n[v1].z - n[v0].z);
                const float factor = sqrtf(ni.x * ni.x + ni.y * ni.y + ni.z * ni.z);
                ni.x /= factor;
                ni.y /= factor;
                ni.z /= factor;
                // create vertex
                const int addr_ = v_.addVertex(ei, ni);
                // map index
                ht_.set(v_addr[e], addr_);
            }
        }
        flag <<= 1;
    }
    // construct triangles
    for (int t = 0; t < 16; t += 3)
    {
        const int index = i_case * 16 + t;
        const int i0 = static_cast<int>(l_tables.t_pattern[index]);
        const int i1 = static_cast<int>(l_tables.t_pattern[index + 1]);
        const int i2 = static_cast<int>(l_tables.t_pattern[index + 2]);
        if (i0 == -1)
            break;
        // add triangle to list
        t_.addTriangle(v_addr[i0], v_addr[i1], v_addr[i2]);
        //t_.addTriangle(ht_.v(v_addr[i0]), ht_.v(v_addr[i1]), ht_.v(v_addr[i2]));
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      SHARED VERTEX or INDEXED FACE DATA STRUCTURES
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Map vertex global id to position in vertex array
// Construct shared vertex list of triangles
__global__ void map_triangles(VertexHashTable ht_, Triangles t_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= t_.nr_t) return;
    const int v0 = t_.triangles[tid].x;
    const int v1 = t_.triangles[tid].y;
    const int v2 = t_.triangles[tid].z;

    t_.triangles[tid].x = ht_.v(v0);
    t_.triangles[tid].y = ht_.v(v1);
    t_.triangles[tid].z = ht_.v(v2);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Map vertex global id to position in vertex array
// Construct shared vertex list for quadrilaterals
__global__ void map_quadrilaterals(QuadrilateralHashTable ht_, Quadrilaterals q_)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= ht_.size()) return;
    // empty bucket
    if (ht_.empty(tid)) return;
    // add quadrilateral
    //q_.addColoredQuadrilateral(ht_.v0(tid), ht_.v1(tid), ht_.v2(tid), ht_.v3(tid), ht_.color(tid));
    q_.addColoredQuadrilateral(ht_.quadrilateral(tid), ht_.color(tid));
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// color elements, inherit coloring from uniform grid, a coloring with 5 colors is always possible
// This method works, because the quad mesh has already a consistent coloring inherited from the uniform grid,
// that is, neighbors have a color different from c.
__global__ void color_quadrilaterals(Quadrilaterals q_, Halfedges he_, HalfedgeFaces he_f, int c)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q) return;
    if (q_.getColor(tid) != c) return; // we are processing color c
    // collect halfedge from face
    const int e0 = he_f.he_e[tid];
    const int e1 = he_.getNext(e0); // he_.he_e[e0].z;
    const int e2 = he_.getNext(e1); // he_.he_e[e1].z;
    const int e3 = he_.getNext(e2); // he_e[e2].z;
    // collect twins
    const int t0 = he_.getTwin(e0); // he_.he_e[e0].w;
    const int t1 = he_.getTwin(e1); // he_.he_e[e1].w;
    const int t2 = he_.getTwin(e2); // he_.he_e[e2].w;
    const int t3 = he_.getTwin(e3); // he_.he_e[e3].w;
    // collect neighboor faces
    int f0{ -1 };
    int f1{ -1 };
    int f2{ -1 };
    int f3{ -1 };
    if (t0 > -1) f0 = he_.getFace(t0); // he_.he_e[t0].y;
    if (t1 > -1) f1 = he_.getFace(t1); // he_.he_e[t1].y;
    if (t2 > -1) f2 = he_.getFace(t2); // he_.he_e[t2].y;
    if (t3 > -1) f3 = he_.getFace(t3); // he_.he_e[t3].y;
    // collect colors
    int c0{ -1 }; if (f0 > -1) c0 = q_.getColor(f0);
    int c1{ -1 }; if (f1 > -1) c1 = q_.getColor(f1);
    int c2{ -1 }; if (f2 > -1) c2 = q_.getColor(f2);
    int c3{ -1 }; if (f3 > -1) c3 = q_.getColor(f3);
    // check which is the first free color from the first 5 colors
    if (c0 != 0 && c1 != 0 && c2 != 0 && c3 != 0)
    {
        q_.setColor(tid, 0);
        return;
    }
    if (c0 != 1 && c1 != 1 && c2 != 1 && c3 != 1)
    {
        q_.setColor(tid, 1);
        return;
    }
    if (c0 != 2 && c1 != 2 && c2 != 2 && c3 != 2)
    {
        q_.setColor(tid, 2);
        return;
    }
    if (c0 != 3 && c1 != 3 && c2 != 3 && c3 != 3)
    {
        q_.setColor(tid, 3);
        return;
    }
    if (c0 != 4 && c1 != 4 && c2 != 4 && c2 != 4)
    {
        q_.setColor(tid, 4);
        return;
    }
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// optimize element colors by removing the fifth color, if possible
// remember halfedge data structure
//    he.x = origin vertex
//    he.y = face
//    he.z = next
//    he.w = twin
__global__ void optimize_coloring(Quadrilaterals q_, Halfedges he_, HalfedgeFaces he_f)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q) return;
    if (q_.getColor(tid) != 4) return; // we are processing color c
    // collect halfedge from face
    const int e0 = he_f.he_e[tid];
    const int e1 = he_.he_e[e0].z;
    const int e2 = he_.he_e[e1].z;
    const int e3 = he_.he_e[e2].z;
    // collect twins
    const int t0 = he_.he_e[e0].w;
    const int t1 = he_.he_e[e1].w;
    const int t2 = he_.he_e[e2].w;
    const int t3 = he_.he_e[e3].w;
    // collect neighboor faces
    int f0{ -1 };
    int f1{ -1 };
    int f2{ -1 };
    int f3{ -1 };
    if (t0 > -1) f0 = he_.he_e[t0].y;
    if (t1 > -1) f1 = he_.he_e[t1].y;
    if (t2 > -1) f2 = he_.he_e[t2].y;
    if (t3 > -1) f3 = he_.he_e[t3].y;
    // collect colors
    int c0{ -1 }; if (f0 > -1) c0 = q_.getColor(f0);
    int c1{ -1 }; if (f1 > -1) c1 = q_.getColor(f1);
    int c2{ -1 }; if (f2 > -1) c2 = q_.getColor(f2);
    int c3{ -1 }; if (f3 > -1) c3 = q_.getColor(f3);
    // check which is the first free color from the first 5 colors
    if (c0 != 0 && c1 != 0 && c2 != 0 && c3 != 0)
    {
        q_.setColor(tid, 0);
        return;
    }
    if (c0 != 1 && c1 != 1 && c2 != 1 && c3 != 1)
    {
        q_.setColor(tid, 1);
        return;
    }
    if (c0 != 2 && c1 != 2 && c2 != 2 && c3 != 2)
    {
        q_.setColor(tid, 2);
        return;
    }
    if (c0 != 3 && c1 != 3 && c2 != 3 && c3 != 3)
    {
        q_.setColor(tid, 3);
        return;
    }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// transfer attributes from quadrilaterals to halfedge faces
__global__ void transfer_face_attributes(Quadrilaterals q_, HalfedgeFaces he_f)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q) return;
    he_f.attributes[tid] = q_.attributes[tid];
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// mark vertices used by quadrilaterals
__global__ void count_unused_vertices(Quadrilaterals q_, bool* flag)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q) return;
    flag[q_.v0(tid)] = true;
    flag[q_.v1(tid)] = true;
    flag[q_.v2(tid)] = true;
    flag[q_.v3(tid)] = true;
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// mark vertices used by quadrilaterals
__global__ void remove_unused_vertices(Vertices v_, Vertices nv_, bool* flag, int* map_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= v_.nr_v) return;
    if (flag[tid])
    {
        const int addr = nv_.addVertex(v_.vertices[tid], v_.normals[tid]);
        map_[tid] = addr;
    }
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// mark vertices used by quadrilaterals
__global__ void remap_quad_indices(Quadrilaterals q_, int* map_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= q_.nr_q) return;
    int index = q_.v0(tid);
    q_.quadrilaterals[tid].x = map_[index];
    index = q_.v1(tid);
    q_.quadrilaterals[tid].y = map_[index];
    index = q_.v2(tid);
    q_.quadrilaterals[tid].z = map_[index];
    index = q_.v3(tid);
    q_.quadrilaterals[tid].w = map_[index];
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// copy from vertices to thrust vectors
__global__ void copy_vertex_coordinates(Vertices v_, float* x_, float* y_, float* z_)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= v_.nr_v) return;
    x_[tid] = v_.x(tid);
    y_[tid] = v_.y(tid);
    z_[tid] = v_.z(tid);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      HOST CODE
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//      Dual Marching Cubes -- Host code
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// remove unused vertices
void removeUnusedVertices(p_mc::Vertices& v_, p_mc::Quadrilaterals& q_)
{
    const int nr_v = v_.size();
    const int nr_q = q_.size();
    p_mc::Vertices nv_(nr_v);

    // set flag for used vertices
    bool* flag{ nullptr };
    hipMalloc(&flag, nr_v * sizeof(bool));
    p_mc::cudaCheckError();
    hipMemset(flag, false, nr_v * sizeof(bool));
    p_mc::cudaCheckError();
    int* map_{ nullptr };
    hipMalloc(&map_, nr_v * sizeof(int));
    p_mc::cudaCheckError();
    hipMemset(map_, INVALID_INDEX, nr_v * sizeof(int));
    // count unused vertices
    int b_size = MC_BLOCKSIZE;
    int g_size = (nr_q + b_size - 1) / b_size;
    count_unused_vertices << < g_size, b_size >> > (q_, flag);
    hipDeviceSynchronize();
    p_mc::cudaCheckError();
    // remove unused vertices
    g_size = (nr_v + b_size - 1) / b_size;
    remove_unused_vertices << <g_size, b_size >> > (v_, nv_, flag, map_);
    // remap indices
    g_size = (nr_q + b_size - 1) / b_size;
    remap_quad_indices << < g_size, b_size >> > (q_, map_);
    // copy vertices back to input array
    v_.copy(nv_);
    // free memory
    hipFree(flag);
    hipFree(map_);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// compute vertex valence distribution to check mesh quality
/// this function is used for measurement purposes
void checkVertexValence(p_mc::Vertices v, p_mc::Quadrilaterals q, std::vector<int>& valenceDist)
{
    /// data
    p_mc::HalfedgeMesh hm;
    p_mc::Edges e;
    p_mc::EdgeHashTable eht;
    /// computes edges
    hm.edgeHashTable(q, eht);
    std::vector<int2> le;
    std::vector<int> nrF;
    eht.getEdges(le);
    eht.getNrFaces(nrF);

    std::vector<int> valence(v.size(), 0);
    for (int i = 0; i < le.size(); i++)
    {
       if (nrF[i] > 0)
        {
            valence[le[i].x] += 1;
            valence[le[i].y] += 1;
        }

    }
    // a unit cell has 12 edges
    // if a single branch of the iso-surface
    // intersects all edge, the max valence might be
    // 12, consider also valence 0
    valenceDist.resize(13, 0);
    for (auto v : valence)
    {
        valenceDist[v] += 1;
    }
}


/// CPU function to control mesh structure
void checkMeshConsistency(p_mc::Vertices v, p_mc::Quadrilaterals q, std::string const &step)
{
    p_mc::HalfedgeMesh hm;
    p_mc::HalfedgeFaces hef;
    p_mc::HalfedgeVertices hev;
    p_mc::Halfedges he;
    p_mc::Edges e;
    p_mc::EdgeHashTable eht;

    /// computes edges
    hm.edgeHashTable(q, eht);
    std::vector<int2> le;
    std::vector<int> nrF;
    eht.getEdges(le);
    eht.getNrFaces(nrF);
    // count nr. of boundary and non-manifold edges
    int nrBndEdges{ 0 };
    int nrInnerEdges{ 0 };
    int nrNonManifold{ 0 };
    int nrWrongEdges{ 0 };
    for (int i = 0; i < le.size(); i++)
    {
        switch (nrF[i])
        {
        case 0:
            break;
        case 1:
            nrBndEdges++;
            break;
        case 2:
            nrInnerEdges++;
            break;
        case 4:
            nrNonManifold++;
            break;
        default:
            nrWrongEdges++;
            break;
        }
    }

    // print
    std::cout << " ... Mesh consistency" << std::endl;
    std::cout << " ... nr. boundary edges: " << nrBndEdges << std::endl;
    std::cout << " ... nr. of manifold inner edges: " << nrInnerEdges << std::endl;
    std::cout << " ... nr. of non-manifold edges: " << nrNonManifold << std::endl;
    std::cout << " ... nr. of WRONG edges: " << nrWrongEdges << std::endl;
    ///// test helfedge mesh
    //CTimer timer;
    //hm.halfedges(v.size(), q, he, hef, hev, timer);
    //// copy data back
    //std::vector<int> lf;
    //std::vector<unsigned char> la;
    //hef.getHalfedgeFaces(lf, la);
    //int nrNonManifoldHalfedgeFaces{ 0 };
    //for (int i = 0; i < la.size(); i++)
    //{
    //    bool flag = la[i] & 0x20;
    //    if (flag) nrNonManifoldHalfedgeFaces++;
    //}
    //std::cout << " ... nr. of non-manifold halfedge faces: " << nrNonManifoldHalfedgeFaces << std::endl;
    //// get quads
    //std::vector<int4> lq;
    //std::vector<unsigned char> lqa;
    //q.getQuadrilaterals(lq, lqa);
    //int nrNonManifoldQuadrilaterals{ 0 };
    //for (int i = 0; i < lqa.size(); i++)
    //{
    //    bool flag = lqa[i] & 0x20;
    //    if (flag) nrNonManifoldQuadrilaterals++;
    //}
    //std::cout << " ... nr. of non-manifold quadrilaterals: " << nrNonManifoldQuadrilaterals << std::endl;
    //// compute vertex valences
    //std::vector<int4> le;
    //he.getHalfedges(le);
    //for (auto e : le)
    //{

    //}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Dual Marching Cubes
void
p_mc::DualMarchingCubes::dualMC(const float i0,
    std::vector<Vertex>& v, std::vector<Normal>& n, std::vector<Triangle>& t, std::vector<Quadrilateral>& q,
    std::vector<Halfedge>& o_he, std::vector<HalfedgeFace>& o_hef, std::vector<HalfedgeVertex>& o_hev,
    std::map<std::string, int>& config)
{
    // CUDA
    uint b_size{ 0 };
    uint g_size{ 0 };
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // configuration
    bool valenceFlag{ static_cast<bool>(config["valence"]) };
    bool elementQualityFlag{ static_cast<bool>(config["element-quality"]) };
    bool p3X3YColor{ static_cast<bool>(config["p3X3YColor"]) };
    bool p3X3YOld{ static_cast<bool>(config["p3X3YOld"]) };
    bool p3333{ static_cast<bool>(config["p3333"]) };
    bool heDataStructure{ static_cast<bool>(config["halfedge-datastructure"]) };
    bool countNonManifolEdges{ static_cast<bool>(config["non-manifold"]) };

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem size
    addElementsInfo(i0);

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem size
    const int t_size = ugrid.t_size(); // dims[0] * dims[1] * dims[2];

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// measure processing time
	CTimer ctimer;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// processing time
    std::cout << " ... compute iso-surface\n";

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// compute quadrilaterals
    // 0. alloc lookup tables
	// 1. alloc memory for hash table
	// 2. alloc memory for vertices
	// 3. alloc memory for quads
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// 0. Cell intersection object
    CellIntersection c_(r_pattern, t_ambig);

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Count number of vertices, which can be used to estimate the number of elements
    ctimer.start();
    int* aCnt{ nullptr };
    hipMalloc(&aCnt, sizeof(int));
    hipMemset(aCnt, 0, sizeof(int));
    b_size = MC_BLOCKSIZE;
    g_size = (t_size + b_size - 1) / b_size;
    count_dmc << < g_size, b_size >> > (i0, t_size, ugrid, c_, aCnt);
    hipDeviceSynchronize();
    cudaCheckError();
    int nrVerts{ 0 };
    hipMemcpy(&nrVerts, aCnt, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckError();
    ctimer.stop();
    std::cout << " ... nr. of estimated vertices " << nrVerts << ", in " << ctimer.getTime() << " ms" << std::endl;

    // We assume there will give a maximum of 15e6 of vertices
    // Hash tables should used the 70% rule of thumb, i.e. multiply size by 100/70
    const int vBuffSize = static_cast<int>(nrVerts); // max number of vertices
    const int qBuffSize = static_cast<int>(1.2 * nrVerts); // assume there will give almost the same nr. of quads as vertices
    const int htBuffSize = static_cast<int>(100. / 70. * qBuffSize);
    // 1. allocate and initialize hash table
	const int ht_size = static_cast<int>(htBuffSize);
    QuadrilateralHashTable ht_(ht_size);
	//cudaCheckError();
	b_size = MC_BLOCKSIZE;
	g_size = (static_cast<uint>(ht_.size()) + b_size - 1) / b_size;
	init_quadrilateral_hashtable << < g_size, b_size >> > (ht_);
    hipDeviceSynchronize();
	cudaCheckError();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// 2. allocate and initialize vertices
    int nr_v{ 0 };
    Vertices v_(vBuffSize);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// 3. allocate and initialize quadrilaterals
    int nr_q{ 0 };
    Quadrilaterals q_(qBuffSize);
    hipDeviceSynchronize();
	cudaCheckError();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// 4. compute iso-surface
    ctimer.start();
	b_size = MC_BLOCKSIZE;
	g_size = (t_size + b_size - 1) / b_size;
    dual_mc << < g_size, b_size >> > (i0, t_size, ugrid, c_, ht_, v_);
	hipDeviceSynchronize();
    cudaCheckError();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// 5. compute shared vertex list for quadrilateral mesh
	// indices of quadrilateral vertices have to be mapped to global vertex index in vertex array
	// get number of vertices
	nr_v = v_.size();
    if (nr_v == 0)
    {
        std::cout << " ERROR: no vertices\n";
        return;
    }

	// map quadrilateral indices
	b_size = MC_BLOCKSIZE;
	g_size = (ht_.size() + b_size - 1) / b_size;
	map_quadrilaterals <<< g_size, b_size >>> (ht_, q_);
    hipDeviceSynchronize();
    cudaCheckError();
	// get number of quadrilaterals
	nr_q = q_.size();

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // compute processing time
    ctimer.stop();
    addTimeInfo(TimeInfo::TimeDMC, ctimer.getTime());
    std::cout << " ... DMC processing time: " << ctimer.getTime() << std::endl;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // remove unused vertices
    removeUnusedVertices(v_, q_);

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // output number of elements
    addElementsInfo(ElementsInfo::ElementsDMC, nr_v, nr_q);


    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Require halfedge data structure
    HalfedgeHashTable et_;
    Halfedges he_;
    HalfedgeFaces he_f;
    HalfedgeVertices he_v;
    HalfedgeMesh he_m;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Mesh properties: count number of non-manifold edges
    if (countNonManifolEdges)
    {
        const int nrNonManifoldEdges = he_m.nonManifold(q_);
        addElementsInfo(ElementsInfo::ElementsNonManifoldEdges, nrNonManifoldEdges);
    }
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // compute number of failed back projections
    nrFailedProjections1 = static_cast<int>(c_.failedProjections1());
    addElementsInfo(ElementsInfo::ElementsFailedProjections1, nrFailedProjections1);
    nrFailedProjections2 = static_cast<int>(c_.failedProjections2());
    addElementsInfo(ElementsInfo::ElementsFailedProjections2, nrFailedProjections2);
    std::cout << " ... number of failed projections at level 1: " << nrFailedProjections1 << std::endl;
    std::cout << " ... number of failed projections at level 2: " << nrFailedProjections2 << std::endl;
    // compute halfedge mesh data structure, processing time is measured within the method
    int nr_e = he_m.halfedges(nr_v, q_, he_, he_f, he_v, ctimer);
    addTimeInfo(TimeInfo::TimeHE, ctimer.getTime());

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Check face coloring method
    std::cout << " ... compute element coloring" << std::endl;
    FaceColoring fc;
    fc.colorFaces(q_, he_, he_f, ctimer);
    ctimer.stop();
    addTimeInfo(TimeInfo::TimeFaceColoring, ctimer.getTime());

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Copy all attributes of quadrilaterals to halfedge faces
    g_size = q_.size();
    transfer_face_attributes << < g_size, b_size >> > (q_, he_f);
    hipDeviceSynchronize();
    cudaCheckError();

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Needs methods in this object for mesh quality measure
    VertexValence vValence;
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Compute the distribution of vertex valence for the DMC mesh
    if (valenceFlag)
    {
        //checkVertexValence(v_, q_, valenceDistDMC);
        vValence.vertexValence(nr_v, q_, valenceDistDMC);
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Remove elements with valence pattern 3-x-3-y and 3-3-3-3
    MeshSimplification ms;
    if (p3X3YColor)
    {
        std::cout << " ... start mesh simplification p3X3Y Color" << std::endl;
        ms.pattern3X3Y(v_, q_, he_, he_f, he_v, ctimer);
        addTimeInfo(TimeInfo::TimeP3X3YColor, ctimer.getTime());
        // re-compute halfedge data structure
        he_m.halfedges(v_.size(), q_, he_, he_f, he_v, ctimer);
        /// compute number of elements
        nr_v = v_.size();
        nr_q = q_.size();
        addElementsInfo(ElementsInfo::ElementsP3X3YColor, nr_v, nr_q);
        /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Compute the distribution of vertex valence for the DMC mesh after P3X3YColor simplification
        if (valenceFlag)
        {
            //checkVertexValence(v_, q_, valenceDistP3X3YColor);
            vValence.vertexValence(nr_v, q_, valenceDistP3X3YColor);
        }
    }
    if (p3X3YOld)
    {
        std::cout << " ... start mesh simplification P3X3Y Old" << std::endl;
        ms.pattern3X3YOld(v_, q_, he_, he_f, he_v, ctimer);
        addTimeInfo(TimeInfo::TimeP3X3YOld, ctimer.getTime());
        // re-compute halfedge data structure
        he_m.halfedges(v_.size(), q_, he_, he_f, he_v, ctimer);
        // compute number of elements
        nr_v = v_.size();
        nr_q = q_.size();
        addElementsInfo(ElementsInfo::ElementsP3X3YOld, nr_v, nr_q);
        /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Compute the distribution of vertex valence for the DMC mesh after P3X3YOld simplification
        if (valenceFlag)
        {
            //checkVertexValence(v_, q_, valenceDistP3X3YOld);
            vValence.vertexValence(nr_v, q_, valenceDistP3X3YOld);
        }
    }
    if (p3333)
    {
        std::cout << " ... start mesh simplification 3333" << std::endl;
        ms.pattern3333(v_, q_, he_, he_f, he_v, ctimer);
        addTimeInfo(TimeInfo::TimeP3333, ctimer.getTime());
        // re-compute halfedge data structure
        he_m.halfedges(v_.size(), q_, he_, he_f, he_v, ctimer);
        nr_v = v_.size();
        nr_q = q_.size();
        addElementsInfo(ElementsInfo::ElementsP3333, nr_v, nr_q);
        /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Compute the distribution of vertex valence for the DMC mesh after P3333 simplification
        if (valenceFlag)
        {
            //checkVertexValence(v_, q_, valenceDistP3333);
            vValence.vertexValence(nr_v, q_, valenceDistP3333);
        }
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Generate a triangle mesh by optimal subdivision of quadrilaterals into triangles
    nr_v = v_.size();
    nr_q = q_.size();
    const int nr_t = 2 * nr_q;
    Triangles t_(nr_t);
    g_size = (static_cast<uint>(t_.a_size) + b_size - 1) / b_size;
    quadrilateral_to_triangle<<< g_size, b_size>>>(q_, v_, t_);
    hipDeviceSynchronize();
    cudaCheckError();
    if (valenceFlag)
    {
        //checkVertexValence(v_, q_, valenceDistP3333);
        vValence.vertexValence(nr_v, t_, valenceDistTris);
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Compute quality of elements, triangles and quadrilaterals
    if (elementQualityFlag)
    {
        EstimateElementQuality eQ;
        ElementQuality eQuads(nr_q);
        ElementQuality eTris(nr_t);
        eQ.q(v_, q_, eQuads);
        eQ.q(v_, t_, eTris);
        eQuads.getQuality(elementQualityQuads);
        eTris.getQuality(elementQualityTris);
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Generate edges from quadrilaterals, this is for rendering purposes
    Edges qe_;
    int nr_qe = he_m.edges(q_, qe_);

    // Resume
    std::cout << " ... Total number of elements generated " << std::endl;
    std::cout << " ... total nr. of vertices " << nr_v << std::endl;
    std::cout << " ... total nr. of quadrilaterals " << nr_q << std::endl;
    std::cout << " ... total nr. of triangles " << nr_t << std::endl;
    std::cout << " ... total nr. of edges " << nr_qe << std::endl;

	// create host mesh
    v.resize(nr_v);
    n.resize(nr_v);
    t.resize(nr_t);
    q.resize(nr_q);
    // copy vertices and normalsData
    float3* v_array = new float3[nr_v];
    float3* n_array = new float3[nr_v];
    hipMemcpy(v_array, v_.vertices, nr_v * sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(n_array, v_.normals, nr_v * sizeof(float3), hipMemcpyDeviceToHost);
	for (int id = 0; id < nr_v; id++) {
		// copy vertices
        v[id] = { v_array[id].x,v_array[id].y, v_array[id].z };
        n[id] = { -n_array[id].x, -n_array[id].y, -n_array[id].z };
	}
    delete[] v_array;
    delete[] n_array;
    // copy mesh elements
    hipMemcpy(q.data(), q_.quadrilaterals, nr_q * sizeof(int4), hipMemcpyDeviceToHost);
    hipMemcpy(t.data(), t_.triangles, nr_t * sizeof(int3), hipMemcpyDeviceToHost);

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // copy half edge data structure from device to host
    if (heDataStructure)
    {
        nr_v = he_v.size(); // number of halfedge vertices
        nr_q = he_f.size(); // number of halfedge faces
        nr_e = he_.size(); // number of halfedge edges
        std::vector<int4> he_e_array;
        he_.getHalfedges(he_e_array);
        // copy to output data structure
        o_he.resize(he_e_array.size());
        for (size_t i = 0; i < he_e_array.size(); i++)
        {
            o_he[i][0] = he_e_array[i].x;
            o_he[i][1] = he_e_array[i].y;
            o_he[i][2] = he_e_array[i].z;
            o_he[i][3] = he_e_array[i].w;
        }
        // get halfedge faces
        std::vector<uchar> he_f_attributes;
        he_f.getHalfedgeFaces(o_hef, he_f_attributes);
        // get halfege Vertices
        he_v.getHalfedgeVertices(o_hev);
    }
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // compute bounding box using thrust
    /*thrust::device_vector<float> x(nr_v);
    thrust::device_vector<float> y(nr_v);
    thrust::device_vector<float> z(nr_v);
    g_size = (static_cast<uint>(nr_v) + b_size - 1) / b_size;
    copy_vertex_coordinates << <g_size, b_size >> > (v_,
            thrust::raw_pointer_cast(x.data()),
            thrust::raw_pointer_cast(y.data()),
            thrust::raw_pointer_cast(z.data()));
    using result_type = thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator>;
    result_type xMinMax = thrust::minmax_element(x.begin(), x.end());
    result_type yMinMax = thrust::minmax_element(y.begin(), y.end());
    result_type zMinMax = thrust::minmax_element(z.begin(), z.end());
    float minX = *xMinMax.first; // x[xMinMax.first - x.begin()];
    float maxX = *xMinMax.second;
    float minY = *yMinMax.first;
    float maxY = *yMinMax.second;
    float minZ = *zMinMax.first;
    float maxZ = *zMinMax.second;*/

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// done!
	std::cout << " ... done\n";
}

void p_mc::DualMarchingCubes::checkFaceColoring(std::vector<int4>& he_e_array, std::vector<int>& he_f_array, std::vector<uchar>& fc_array)
{
    int cnt{ 0 };
    int a0{ 0 };
    int a1{ 0 };
    int a2{ 0 };
    int a3{ 0 };
    int a4{ 0 };
    int cA{ 0 };
    int cSimpl{ 0 };
    auto getColor = [](uchar c) { return static_cast<int>(c&0x1F); };
    // loop over all quadrilaterals
    const int nr_q = static_cast<int>(he_f_array.size());
    for (int f = 0; f < nr_q; f++)
    {
        // collect halfedge
        const int e0 = he_f_array[f];
        const int e1 = he_e_array[e0].z;
        const int e2 = he_e_array[e1].z;
        const int e3 = he_e_array[e2].z;
        // collect all four twin edges
        const int t0 = he_e_array[e0].w;
        const int t1 = he_e_array[e1].w;
        const int t2 = he_e_array[e2].w;
        const int t3 = he_e_array[e3].w;
        // collect faces, if any
        int f0{ -1 };
        int f1{ -1 };
        int f2{ -1 };
        int f3{ -1 };
        if (t0 > -1) f0 = he_e_array[t0].y;
        if (t1 > -1) f1 = he_e_array[t1].y;
        if (t2 > -1) f2 = he_e_array[t2].y;
        if (t3 > -1) f3 = he_e_array[t3].y;
        // collect colors
        int c = getColor(fc_array[f]);
        int c0{ -1 };
        int c1{ -1 };
        int c2{ -1 };
        int c3{ -1 };
        if (f0 > -1) c0 = getColor(fc_array[f0]);
        if (f1 > -1) c1 = getColor(fc_array[f1]);
        if (f2 > -1) c2 = getColor(fc_array[f2]);
        if (f3 > -1) c3 = getColor(fc_array[f3]);
        // compare with own color
        bool flag{ false };
        if (c == c0) flag = true;
        if (c == c1) flag = true;
        if (c == c2) flag = true;
        if (c == c3) flag = true;
        if (c == 0) a0++;
        if (c == 1) a1++;
        if (c == 2) a2++;
        if (c == 3) a3++;
        if (c == 4) a4++;
        if (c > 4) cA++;
        if (c == 4)
        {
            int k{ 0 };
            if (c0 == 0 || c1 == 0 || c2 == 0 || c3 == 0) k++;
            if (c0 == 1 || c1 == 1 || c2 == 1 || c3 == 1) k++;
            if (c0 == 2 || c1 == 2 || c2 == 2 || c3 == 2) k++;
            if (c0 == 3 || c1 == 3 || c2 == 3 || c3 == 3) k++;
            if (k == 4) cSimpl++;
        }
        if (flag)
        {
            //std::string msg = "ERROR for quadrilateral " + std::to_string(f) + "\n";
            //d_out::print(msg);
            cnt++;
        }
    }
    std::cout << " ... ERROR: " << cnt << " faces do not comply coloring condition" << std::endl;
    std::cout << " ... Color 0: " << a0 << " faces have color 0" << std::endl;
    std::cout << " ... Color 1: " << a1 << " faces have color 1" << std::endl;
    std::cout << " ... Color 2: " << a2 << " faces have color 2" << std::endl;
    std::cout << " ... Color 3: " << a3 << " faces have color 3" << std::endl;
    std::cout << " ... Color 4: " << a4 << " faces have color 4" << std::endl;
    std::cout << " ... Color A: " << cA << " faces have color A" << std::endl;
    std::cout << " ... Color complete: " << cSimpl << " faces have all four colors as neighbors" << std::endl;
}

void p_mc::DualMarchingCubes::checkHalfedge(std::vector<int4>& he_e_array, std::vector<int>& he_f_array, std::vector<Quadrilateral>& quads)
{
    /** halfedge int4:
    //    he.x = origin vertex
    //    he.y = face
    //    he.z = next
    //    he.w = twin
    */
    std::map<int, int> m_;
    int c{ 0 };
    const int nr_e = static_cast<int>(he_e_array.size());
    for (int e = 0; e < nr_e; e++)
    {
        int4 he0 = he_e_array[e];
        // construct quadrilateral, compare with quads from mesh.
        const int ihe0 = e;
        const int ihe1 = he0.z;
        int4 he1 = he_e_array[ihe1];
        const int ihe2 = he1.z;
        int4 he2 = he_e_array[ihe2];
        const int ihe3 = he2.z;
        int4 he3 = he_e_array[ihe3];
        int4 heTest = he_e_array[ihe3];
        const int f = he0.y;
        Quadrilateral q = quads[f];
        const int v0 = he0.x;
        const int v1 = he1.x;
        const int v2 = he2.x;
        const int v3 = he3.x;
        // compare vertices
        if (q[0] != v0 && q[0] != v1 && q[0] != v2 && q[0] != v3)
        {
            std::cout << "ERROR: can't find v0\n";
        }
        if (q[1] != v0 && q[1] != v1 && q[1] != v2 && q[1] != v3)
        {
            std::cout << "ERROR: can't find v1\n";
        }
        if (q[2] != v0 && q[2] != v1 && q[2] != v2 && q[2] != v3)
        {
            std::cout << "ERROR: can't find v2\n";
        }
        if (q[3] != v0 && q[3] != v1 && q[3] != v2 && q[3] != v3)
        {
            std::cout << "ERROR: can't find v3\n";
        }

        // check twins
        const int t0 = he0.w;
        const int t1 = he1.w;
        const int t2 = he2.w;
        const int t3 = he3.w;
        int tv0{ -1 };
        int tv1{ -1 };
        int tv2{ -1 };
        int tv3{ -1 };
        if (t0 != -1)
        {
            tv0 = he_e_array[t0].w;
            int ni = he_e_array[t0].z;
            int4 ne = he_e_array[ni];
            if (tv0 != ihe0)
            {
                std::cout << "ERROR: wrong twin for he 0\n";
            }
            if (v0 != ne.x)
            {
                std::cout << "ERROR: wrong vertex config by twin edges\n";
            }

        }
        if (t1 != -1)
        {
            tv1 = he_e_array[t1].w;
            if (tv1 != ihe1)
            {
                std::cout << "ERROR: wrong twin for he 1\n";
            }

        }
        if (t2 != -1)
        {
            tv2 = he_e_array[t2].w;
            if (tv2 != ihe2)
            {
                std::cout << "ERROR: wrong twin for he 0\n";
            }

        }
        if (t3 != -1)
        {
            tv3 = he_e_array[t3].w;
            if (tv3 != ihe3)
            {
                std::cout << "ERROR: wrong twin for he 0\n";
            }

        }


    }
    std::cout << " ... nr. of bnd edges: " << c << std::endl;
    return;
    // find edges
    std::vector<std::array<int, 2>> edges;
    for (auto e : m_)
    {
        const int n = he_e_array[e.second].z;
        const int vn = he_e_array[n].x;
        // check if vertex is in list
        auto search = m_.find(vn);
        if (search != m_.end()) {
            //std::cout << "Found " << search->first << " " << search->second << '\n';
            edges.push_back({ e.first,vn });
        }
        else {
            std::cout << "Not found\n";
        }
    }
    std::cout << "found: " << edges.size() << ", edges" << std::endl;
    // clear double cases
    auto  setKey = [](const int v0, const int v1)
    {
        if (v0 < v1)
            return (static_cast<unsigned long long>(v0) << 32) | (v1 & 0xffffffffL);
        else
            return (static_cast<unsigned long long>(v1) << 32) | (v0 & 0xffffffffL);
    };
    std::map<unsigned long long, std::array<int, 2>> em_;
    for (auto e : edges)
    {
        em_.insert({setKey(e[0] ,e[1]),e});
    }
    std::cout << "after clear: " << em_.size() << ", edges" << std::endl;
    // analyze cases
    /*std::ofstream f;
    f.open("./data/models/nonManifold.txt");
    auto boundary = [](const int i, const int j, const int k, UGrid& u)
    {
        bool flag{ false };
        if (i == 0 || i == u.i_size()) flag = true;
        if (j == 0 || j == u.j_size()) flag = true;
        if (k == 0 || k == u.k_size()) flag = true;
        return flag;
    };
    auto scalar = [](const int i_, const int j_, const int k_, float* v, UGrid& u)
    {
        std::array<double, 8> s;
        for (int k = 0; k < 2; k++)
        {
            for (int j = 0; j < 2; j++)
            {
                for (int i = 0; i < 2; i++)
                {
                    int index = (i & 1) | (j & 1) << 1 | (k & 1) << 2;
                    int gl = u.gl_index(i_ + i, j_ + j, k_ + k);
                    s[index] = v[gl];
                }
            }
        }
        return s;
    };*/
    //for (auto e : em_)
    //{
    //    const int v0 = e.second[0];
    //    const int v1 = e.second[1];
    //    const int c0 = info[v0].x;
    //    const int c1 = info[v1].x;
    //    const int g0 = info[v0].z;
    //    const int g1 = info[v1].z;
    //    const int i0 = ugrid.i_index(g0);
    //    const int j0 = ugrid.j_index(g0);
    //    const int k0 = ugrid.k_index(g0);
    //    const int i1 = ugrid.i_index(g1);
    //    const int j1 = ugrid.j_index(g1);
    //    const int k1 = ugrid.k_index(g1);
    //    if (!boundary(i0, j0, k0, ugrid) && !boundary(i1, j1, k1, ugrid))
    //    {
    //        auto s0 = scalar(i0, j0, k0, h_volume, ugrid);
    //        auto s1 = scalar(i1, j1, k1, h_volume, ugrid);
    //        f << s0[0] << "," << s0[1] << "," << s0[2] << "," << s0[3] << "," << s0[4] << "," << s0[5] << "," << s0[6] << "," << s0[7] << "," << i0 << "," << j0 << "," << k0 << std::endl;
    //        f << s1[0] << "," << s1[1] << "," << s1[2] << "," << s1[3] << "," << s1[4] << "," << s1[5] << "," << s1[6] << "," << s1[7] << "," << i1 << "," << j1 << "," << k1 << std::endl;
    //    }


    //    //std::cout << "(" << c0 << ", " << i0 << ", " << j0 << ", " << k0 << "), (" << c1 << ", " << i1 << ", " << j1 << ", " << k1 << "), " << std::endl;
    //}
    //f.close();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Write infos to files for measurement purposes
void p_mc::DualMarchingCubes::writeInfos(const std::string& dataset)
{
    std::ofstream o_file;
    // print time info
    std::string filename = dataset + "_TimeInfo" + ".txt";
    o_file.open(filename);
    for (auto s : timesInfo)
    {
        o_file << s << " ms" << std::endl;
    }
    o_file.close();
    // write element info
    filename = dataset + "_Elements" + ".txt";
    o_file.open(filename);
    for (auto s : elementsInfo)
    {
        o_file << s  << std::endl;
    }
    o_file.close();
    // valence distribution DMC
    filename = dataset + "_valenceDMC" + ".txt";
    o_file.open(filename);
    for (auto s : valenceDistDMC)
    {
        o_file << s << std::endl;
    }
    o_file.close();
    // valence distribution P3X3YColor
    filename = dataset + "_valenceP3X3YColor" + ".txt";
    o_file.open(filename);
    for (auto s : valenceDistP3X3YColor)
    {
        o_file << s << std::endl;
    }
    o_file.close();
    filename = dataset + "_valenceP3X3YOld" + ".txt";
    o_file.open(filename);
    for (auto s : valenceDistDMC)
    {
        o_file << s << std::endl;
    }
    o_file.close();
    filename = dataset + "_valenceP3333" + ".txt";
    o_file.open(filename);
    for (auto s : valenceDistP3333)
    {
        o_file << s << std::endl;
    }
    o_file.close();
    filename = dataset + "_valenceTris" + ".txt";
    o_file.open(filename);
    for (auto s : valenceDistTris)
    {
        o_file << s << std::endl;
    }
    o_file.close();
    // element quality quads
    filename = dataset + "_qualityQuads" + ".bin";
    o_file.open(filename, std::ios::binary);
    int sz = static_cast<int>(elementQualityQuads.size());
    o_file.write(reinterpret_cast<char*>(&sz), sizeof(int));
    o_file.write(reinterpret_cast<char*>(elementQualityQuads.data()), sz * sizeof(float));
    o_file.close();
    filename = dataset + "_qualityTris" + ".bin";
    o_file.open(filename, std::ios::binary);
    sz = static_cast<int>(elementQualityTris.size());
    o_file.write(reinterpret_cast<char*>(&sz), sizeof(int));
    o_file.write(reinterpret_cast<char*>(elementQualityTris.data()), sz * sizeof(float));
    o_file.close();
}
